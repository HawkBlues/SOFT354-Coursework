#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>

#include "win-gettimeofday.h"

/* Number of threads per block */
#define THREADS_PER_BLOCK 512


__global__ void mergeSortLeft(int *array, int *Left, int *Right, int Middle, int End) {
	for (int i = 0; i <= Middle; i++) { //Copy half of the array into the Left
		Left[i] = array[i];
	}
}

__global__ void mergeSortRight(int *array, int *Left, int *Right, int Middle, int End) {
	for (int i = (Middle + 1); i <= End; i++) {
		Right[i] = array[i];  //Copy the second half of the array into the Right
	}
}

__global__ void mergeBoth(int *array, int *Left, int *Right, int Middle, int End) {
	int i = 0;
	int j = 0;
	int k = 0;

	while (i < Middle && j < End) {
	if (Left[i] <= Right[j]) {
		array[k] = Right[j];
		j++;
	}
	else {
		array[k] = Left[i];
		i++;
	}
	k++;
}
}

void populateRandomArray(int *x, int num_elements) {
	for (int i = 0; i < num_elements; i++) {
		x[i] = rand() % 100 + 1;
	}
}




/* In C, the "main" function is treated the same as every function,
*  it has a return type (and in some cases accepts inputs via parameters).
*  The only difference is that the main function is "called" by the operating
*  system when the user runs the program.
*  Thus the main function is always the first code executed when a program starts.
*  This function returns an integer representing the application software status.
*/
int main(void)
{
	const int number_of_trials = 20;


	int trials[number_of_trials];

	int* host_a; //used to store the whole 1d matrix
	int* host_Left; //Used to store half of the matrix
	int* host_Right; //Used to store the other half of the matrix

	int* host_c;//TESTING

	int* device_a;
	int* device_Left;
	int* device_Right;

	int* device_c;//TESTING


	//for (int i = 0; i < number_of_trials; i++) {
	for (int i = 0; i < 1; i++) {
		int size = trials[i] * sizeof(int);

		int middle = (number_of_trials / 2); //Used to find the middle of the matrix
		int end = number_of_trials;//Used to find the end of the matrix

		host_a = (int *)malloc(size);//Used to store the while 1d matrix
		host_Left = (int *)malloc(size);//Used to get the si
		host_Right = (int *)malloc(size);//Used to store the right half of the matrix

		host_c = (int *)malloc(size);//TESTING

		hipMalloc((void **)&device_a, size);
		hipMalloc((void **)&device_Left, (size));
		hipMalloc((void **)&device_Right, (size));
		hipMalloc((void **)&device_c, (size)); //TESTING

		populateRandomArray(host_a, number_of_trials);



		hipMemcpy(device_a, host_a, size, hipMemcpyHostToDevice);
		hipMemcpy(device_Left, host_Left, size, hipMemcpyHostToDevice);
		hipMemcpy(device_Right, host_Right, size, hipMemcpyHostToDevice);

		dim3 dimBlock(THREADS_PER_BLOCK, 1, 1);
		dim3 dimGrid((trials[i] + dimBlock.x - 1) / dimBlock.x, 1, 1);

		printf("Entire Sorted List");
		for (int i = 0; i < number_of_trials; i++) {
			printf("%d,", host_a[i]);
		}
		printf("\n");
	

		mergeSortLeft << < dimGrid, dimBlock >> > (device_a, device_Left, device_Right, middle, end);
		hipDeviceSynchronize();
		mergeSortRight << < dimGrid, dimBlock >> > (device_a, device_Left, device_Right, middle, end);
		hipDeviceSynchronize();
		//mergeBoth << < dimGrid, dimBlock >> > (device_c, device_Left, device_Right, middle, end);
		//When LEft and right are run, they populate device_left and right correctly. This is copied correctly.
		//Yet when trying to copy DeviceC (used for testing) from mergeBoth to see the contents, it's totally blank?

		hipError_t error = hipGetLastError();
		if (error != hipSuccess)
		{
			/* Returns the description string for an error code */
			printf("Error: %s\n", hipGetErrorString(error));
		}

		hipDeviceSynchronize();

		hipMemcpy(host_c, device_c, size, hipMemcpyDeviceToHost);
	

		printf("Entire Sorted List");
		for (int i = 0; i < number_of_trials; i++) {
			printf("%d,", host_c[i]);
		}
		printf("\n");


	
		free(host_a);
		free(host_Right);
		free(host_Left);
		free(host_c); //TESTING

		hipFree(device_a);
		hipFree(device_Left);
		hipFree(device_Right);
		hipFree(device_c);//TESTING

		hipDeviceReset();
	}
	return 0;
}
